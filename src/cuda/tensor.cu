#include "auto_engine/base/exit_code.h"
#include "auto_engine/cuda/info.h"
#include "auto_engine/cuda/mem.h"
#include "auto_engine/cuda/tensor.h"
#include "auto_engine/cuda/kernel.cuh"
#include "auto_engine/utils/defer.h"
#include "hipblas.h"
#include <cstdlib>
#include <tuple>

namespace cuda {

#define CHECK_MALLOC(call) \
{ \
    auto succ = call; \
    if (!succ) { \
        LOG(ERROR) << __FUNCTION__ << " malloc cuda err"; \
        exit(CUDA_ERR); \
    } \
} \

std::tuple<dim3, dim3> get_apply_dims(int size) {
    int grid_dim = (size + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
    return std::tuple(dim3(grid_dim), dim3(sqrt_tcnt_per_block())); 
}

std::tuple<dim3, dim3> get_transpose_dims(int row, int col, int size) {
    int grid_dim_x = (row + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
    int grid_dim_y = (col + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
    int grid_dim_z = size;
    return std::tuple(dim3(grid_dim_x, grid_dim_y, grid_dim_z), dim3(sqrt_tcnt_per_block(), sqrt_tcnt_per_block()));
}

#define DEFINE_APPLY_1E(fn, T) \
void fn(T* data, int size) { \
    f64* m; \
    CHECK_MALLOC(Mem::malloc((void**)&m, sizeof(T) * size)); \
    utils::Defer free_m([&m]() {Mem::free(m);}); \
    CHECK_CUDA_CALL(hipMemcpy(m, data, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    auto dims = get_apply_dims(size); \
    cuda_kernel::fn<<<std::get<0>(dims), std::get<1>(dims)>>>(m, size); \
    CHECK_CUDA_CALL(hipPeekAtLastError(), #fn); \
    CHECK_CUDA_CALL(hipMemcpy(data, m, sizeof(T) * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h"); \
} \

DEFINE_APPLY_1E(apply_sin, f64)
DEFINE_APPLY_1E(apply_cos, f64)
DEFINE_APPLY_1E(apply_log, f64)
DEFINE_APPLY_1E(apply_neg, f64)

#define DEFINE_APPLY_2E(fn, T) \
void fn(T* data1, const T* data2, int size) { \
    f64 *m1, *m2; \
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(T) * size)); \
    utils::Defer free_m1([&m1]() {Mem::free(m1);}); \
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(T) * size)); \
    utils::Defer free_m2([&m2]() {Mem::free(m2);}); \
 \
    CHECK_CUDA_CALL(hipMemcpy(m1, data1, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    CHECK_CUDA_CALL(hipMemcpy(m2, data2, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    auto dims = get_apply_dims(size); \
    cuda_kernel::fn<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, m2, size); \
    CHECK_CUDA_CALL(hipPeekAtLastError(), #fn); \
    CHECK_CUDA_CALL(hipMemcpy(data1, m1, sizeof(T) * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h"); \
} \

DEFINE_APPLY_2E(apply_add, f64)
DEFINE_APPLY_2E(apply_sub, f64)
DEFINE_APPLY_2E(apply_mul, f64)
DEFINE_APPLY_2E(apply_div, f64)
DEFINE_APPLY_2E(apply_pow, f64)

void apply_sum(const f64* src, int src_size, f64* dst, int dst_size) {
    f64 *m1, *m2;
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(f64) * src_size));
    utils::Defer free_m1([&m1]() {Mem::free(m1);});
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(f64) * dst_size));
    utils::Defer free_m2([&m2]() {Mem::free(m2);});

    CHECK_CUDA_CALL(hipMemcpy(m1, src, sizeof(f64) * src_size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    auto dims = get_apply_dims(dst_size);
    cuda_kernel::apply_sum<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, src_size, m2, dst_size);
    CHECK_CUDA_CALL(hipPeekAtLastError(), "apply_sum");
    CHECK_CUDA_CALL(hipMemcpy(dst, m2, sizeof(f64) * dst_size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

void apply_sum(const f64* src, int size, f64* dst) {
    f64 *m1, *m2;
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(f64) * size));
    utils::Defer free_m1([&m1]() {Mem::free(m1);});
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(f64)));
    utils::Defer free_m2([&m2]() {Mem::free(m2);});

    CHECK_CUDA_CALL(hipMemcpy(m1, src, sizeof(f64) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(m2, dst, sizeof(f64), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    auto dims = get_apply_dims(size);
    cuda_kernel::apply_sum<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, size, m2);
    CHECK_CUDA_CALL(hipPeekAtLastError(), "apply_sum");
    CHECK_CUDA_CALL(hipMemcpy(dst, m2, sizeof(f64), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

void transpose(f64* ms, int row, int col, int size) {
    int cnt = row * col * size;
    f64 *m1, *m2;
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(f64) * cnt));
    utils::Defer free_m1([&m1]() {Mem::free(m1);});
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(f64) * cnt));
    utils::Defer free_m2([&m2]() {Mem::free(m2);});

    CHECK_CUDA_CALL(hipMemcpy(m1, ms, sizeof(f64) * cnt, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    auto dims = get_transpose_dims(row, col, size);
    cuda_kernel::transpose<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, m2, row, col, size);
    CHECK_CUDA_CALL(hipPeekAtLastError(), "transpose");
    CHECK_CUDA_CALL(hipMemcpy(ms, m2, sizeof(f64) * cnt, hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

void mmul(int m, int n, int k, const f64* data1, const f64* data2, f64* dst, int size) {
    hipblasHandle_t handle; 
    CHECK_CUBLAS_CALL(hipblasCreate(&handle), "create"); 
    utils::Defer destroy_handle([&handle]() {hipblasDestroy(handle);});

    f64 *cm, **icm, **hicm;
    CHECK_MALLOC(Mem::malloc((void**)&cm, (m * k + k * n + m * n) * size * sizeof(f64)));
    utils::Defer destroy_cm([&cm]() {Mem::free(cm);});
    CHECK_MALLOC(Mem::malloc((void**)&icm, 3 * size * sizeof(f64*)));
    utils::Defer destroy_icm([&icm]() {Mem::free(icm);});
    hicm = (f64**)malloc(3 * size * sizeof(f64*));
    utils::Defer destroy_hicm([&hicm]() {free(hicm);});

    CHECK_CUDA_CALL(hipMemcpy(cm, data1, sizeof(f64) * m * k * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(cm + m * k * size, data2, sizeof(f64) * k * n * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    // dst不需要拷贝

    for (int i = 0; i < size; i++) {
        hicm[i] = cm + i * m * k;
        hicm[i + size] = cm + size * m * k + i * k * n;
        hicm[i + 2 * size] = cm + size * m * k + size * k * n + i * m * n;
    }
    CHECK_CUDA_CALL(hipMemcpy(icm, hicm, 3 * size * sizeof(f64*), hipMemcpyHostToDevice), "cuda_memcpy_h2d");

    const f64 alpha = 1.0, beta = 0.0; 
    CHECK_CUBLAS_CALL(hipblasDgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, icm + size, n, icm, k, &beta, icm + 2 * size, n, size), "gemm_batched");

    CHECK_CUDA_CALL(hipMemcpy(dst, cm + m * k * size + k * n * size, sizeof(f64) * m * n * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

bool inv(int m, f64* data, int size) {
    hipblasHandle_t handle; 
    CHECK_CUBLAS_CALL(hipblasCreate(&handle), "create"); 
    utils::Defer destroy_handle([&handle]() {hipblasDestroy(handle);});

    f64 *cm, **icm, **hicm;
    CHECK_MALLOC(Mem::malloc((void**)&cm, 2 * m * m * size * sizeof(f64)));
    utils::Defer destroy_cm([&cm]() {Mem::free(cm);});
    CHECK_MALLOC(Mem::malloc((void**)&icm, 2 * size * sizeof(f64*)));
    utils::Defer destroy_icm([&icm]() {Mem::free(icm);});
    hicm = (f64**)malloc(2 * size * sizeof(f64*));
    utils::Defer destroy_hicm([&hicm]() {free(hicm);});

    CHECK_CUDA_CALL(hipMemcpy(cm, data, sizeof(f64) * m * m * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    for (int i = 0; i < 2 * size; i++) {
        hicm[i] = cm + i * m * m;
    }
    CHECK_CUDA_CALL(hipMemcpy(icm, hicm, 2 * size * sizeof(f64*), hipMemcpyHostToDevice), "cuda_memcpy_h2d");

    int *pivot_arr, *info_arr, *hinfo_arr;
    CHECK_MALLOC(Mem::malloc((void**)&pivot_arr, m * size * sizeof(int)));
    utils::Defer destroy_pivot_arr([&pivot_arr]() {Mem::free(pivot_arr);});
    CHECK_MALLOC(Mem::malloc((void**)&info_arr, size * sizeof(int)));
    utils::Defer destroy_info_arr([&info_arr]() {Mem::free(info_arr);});
    hinfo_arr = (int*)malloc(size * sizeof(int));
    utils::Defer destroy_hinfo_arr([&hinfo_arr]() {free(hinfo_arr);});

    CHECK_CUBLAS_CALL(hipblasDgetrfBatched(handle, m, icm, m, pivot_arr, info_arr, size), "getrf_batched");

    auto check_info = [&hinfo_arr, &info_arr, &size](const std::string& fn) -> bool {
        CHECK_CUDA_CALL(hipMemcpy(hinfo_arr, info_arr, size * sizeof(int), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
        for (int i = 0; i < size; i++) {
            if (hinfo_arr[i] != 0) {
                LOG(ERROR) << __FUNCTION__ << " check info err: " << fn << " index: " << i << " code: " << hinfo_arr[i];
                return false;
            }
        }
        return true;
    };
    if (!check_info("getrf_batched")) {
        return false;
    }

    CHECK_CUBLAS_CALL(hipblasDgetriBatched(handle, m, icm, m, pivot_arr, icm + size, m, info_arr, size), "dgetri_batched");
    if (!check_info("getri_batched")) {
        return false;
    }

    CHECK_CUDA_CALL(hipMemcpy(data, cm + m * m * size, m * m * size * sizeof(f64), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
    return true;
}




}