#include "hip/hip_runtime.h"
#include "auto_engine/base/exit_code.h"
#include "auto_engine/cuda/info.h"
#include "auto_engine/cuda/mem.h"
#include "auto_engine/cuda/tensor.h"
#include "auto_engine/cuda/kernel.cuh"
#include "auto_engine/shape/shape.h"
#include "auto_engine/utils/defer.h"
#include "hipblas.h"
#include <cstdlib>
#include <fmt/core.h>
#include <tuple>
#include <utility>
#include <vector>

namespace cuda {

#define CHECK_MALLOC(call) \
{ \
    auto succ = call; \
    if (!succ) { \
        LOG(ERROR) << __FUNCTION__ << " malloc cuda err"; \
        exit(CUDA_ERR); \
    } \
} \

std::tuple<dim3, dim3> get_apply_dims(u32 size) {
    u32 grid_dim = (size + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
    return std::tuple(dim3(grid_dim), dim3(sqrt_tcnt_per_block())); 
}

std::tuple<dim3, dim3> get_matrix_dims(u32 row, u32 col, u32 size) {
    u32 grid_dim_x = (row + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
    u32 grid_dim_y = (col + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
    u32 grid_dim_z = size;
    return std::tuple(dim3(grid_dim_x, grid_dim_y, grid_dim_z), dim3(sqrt_tcnt_per_block(), sqrt_tcnt_per_block()));
}

#define DEFINE_APPLY_1E(fn, T) \
void fn(T* data, u32 size) { \
    f64* m; \
    CHECK_MALLOC(Mem::malloc((void**)&m, sizeof(T) * size)); \
    utils::Defer free_m([&m]() {Mem::free(m);}); \
    CHECK_CUDA_CALL(hipMemcpy(m, data, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    auto dims = get_apply_dims(size); \
    cuda_kernel::fn<<<std::get<0>(dims), std::get<1>(dims)>>>(m, size); \
    CHECK_CUDA_CALL(hipPeekAtLastError(), #fn); \
    CHECK_CUDA_CALL(hipMemcpy(data, m, sizeof(T) * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h"); \
} \

DEFINE_APPLY_1E(apply_sin, f64)
DEFINE_APPLY_1E(apply_cos, f64)
DEFINE_APPLY_1E(apply_log, f64)
DEFINE_APPLY_1E(apply_neg, f64)
DEFINE_APPLY_1E(apply_sign, f64)
DEFINE_APPLY_1E(apply_abs, f64)

#define DEFINE_APPLY_2E(fn, T) \
void fn(T* data1, const T* data2, u32 size) { \
    f64 *m1, *m2; \
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(T) * size)); \
    utils::Defer free_m1([&m1]() {Mem::free(m1);}); \
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(T) * size)); \
    utils::Defer free_m2([&m2]() {Mem::free(m2);}); \
 \
    CHECK_CUDA_CALL(hipMemcpy(m1, data1, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    CHECK_CUDA_CALL(hipMemcpy(m2, data2, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    auto dims = get_apply_dims(size); \
    cuda_kernel::fn<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, m2, size); \
    CHECK_CUDA_CALL(hipPeekAtLastError(), #fn); \
    CHECK_CUDA_CALL(hipMemcpy(data1, m1, sizeof(T) * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h"); \
} \

#define DEFINE_APPLY_1E_1T(fn, T) \
void fn(T* data, T n, u32 size) { \
    f64 *m; \
    CHECK_MALLOC(Mem::malloc((void**)&m, sizeof(T) * size)); \
    utils::Defer free_m([&m]() {Mem::free(m);}); \
 \
    CHECK_CUDA_CALL(hipMemcpy(m, data, sizeof(T) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d"); \
    auto dims = get_apply_dims(size); \
    cuda_kernel::fn<<<std::get<0>(dims), std::get<1>(dims)>>>(m, n, size); \
    CHECK_CUDA_CALL(hipPeekAtLastError(), #fn); \
    CHECK_CUDA_CALL(hipMemcpy(data, m, sizeof(T) * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h"); \
} \

DEFINE_APPLY_2E(apply_add, f64)
DEFINE_APPLY_2E(apply_sub, f64)
DEFINE_APPLY_2E(apply_mul, f64)
DEFINE_APPLY_2E(apply_div, f64)
DEFINE_APPLY_2E(apply_pow, f64)

DEFINE_APPLY_1E_1T(apply_add, f64)
DEFINE_APPLY_1E_1T(apply_sub, f64)
DEFINE_APPLY_1E_1T(apply_mul, f64)
DEFINE_APPLY_1E_1T(apply_div, f64)
DEFINE_APPLY_1E_1T(apply_pow, f64)

void sum(const f64* src, u32 size, f64* dst) {
    f64 *m1, *m2;
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(f64) * size));
    utils::Defer free_m1([&m1]() {Mem::free(m1);});
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(f64)));
    utils::Defer free_m2([&m2]() {Mem::free(m2);});

    CHECK_CUDA_CALL(hipMemcpy(m1, src, sizeof(f64) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemset(m2, 0, sizeof(f64)), "cuda_memset");
    auto dims = get_apply_dims(size);
    cuda_kernel::sum<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, size, m2);
    CHECK_CUDA_CALL(hipPeekAtLastError(), "apply_sum");
    CHECK_CUDA_CALL(hipMemcpy(dst, m2, sizeof(f64), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

void expand(const f64* src, f64* dst, u32 size) {
    f64 *m1, *m2;
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(f64)));
    utils::Defer free_m1([&m1]() {Mem::free(m1);});
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(f64) * size));
    utils::Defer free_m2([&m2]() {Mem::free(m2);});

    CHECK_CUDA_CALL(hipMemcpy(m1, src, sizeof(f64), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    auto dims = get_apply_dims(size);
    cuda_kernel::expand<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, m2, size);
    CHECK_CUDA_CALL(hipPeekAtLastError(), "apply_expand");
    CHECK_CUDA_CALL(hipMemcpy(dst, m2, sizeof(f64) * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

int ont_hot(const f64* src, u32 size, f64* dst, u32 classes) {
    int err_index = -1;

    f64 *m1, *m2; u32 *m3;
    CHECK_MALLOC(Mem::malloc((void**)&m1, sizeof(f64) * size));
    utils::Defer free_m1([&m1]() {Mem::free(m1);});
    CHECK_MALLOC(Mem::malloc((void**)&m2, sizeof(f64) * size * classes));
    utils::Defer free_m2([&m2]() {Mem::free(m2);});
    CHECK_MALLOC(Mem::malloc((void**)&m3, sizeof(int)));
    utils::Defer free_m3([&m3]() {Mem::free(m3);});

    CHECK_CUDA_CALL(hipMemcpy(m1, src, sizeof(f64) * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(m3, &err_index, sizeof(int), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    auto dims = get_apply_dims(size);
    cuda_kernel::one_hot<<<std::get<0>(dims), std::get<1>(dims)>>>(m1, size, m2, classes, m3);
    CHECK_CUDA_CALL(hipPeekAtLastError(), "one_hot");
    CHECK_CUDA_CALL(hipMemcpy(&err_index, m3, sizeof(int), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
    if (err_index >= 0) {
        return err_index;
    }
    CHECK_CUDA_CALL(hipMemcpy(dst, m2, sizeof(f64) * size * classes, hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
    return err_index;
}


void sum(const f64* src, f64* dst, const base::Shape& shape, u32 d) {
// __global__ void sum(const T* src, T* dst, const u32* dims, const u32* strides, u32 dim_cnt) {
    f64 *msrc, *mdst;
    u32 *mds;
    CHECK_MALLOC(Mem::malloc((void**)&msrc, sizeof(f64) * shape.tensorSize()));
    utils::Defer free_msrc([&msrc]() {Mem::free(msrc);});
    CHECK_MALLOC(Mem::malloc((void**)&mdst, sizeof(f64) * shape.tensorSize() / shape.getDim(d)));
    utils::Defer free_mdst([&mdst]() {Mem::free(mdst);});
    CHECK_MALLOC(Mem::malloc((void**)&mds, sizeof(u32) * shape.dimCnt() * 2));
    utils::Defer free_mds([&mds]() {Mem::free(mds);});

    CHECK_CUDA_CALL(hipMemcpy(msrc, src, sizeof(f64) * shape.tensorSize(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(mds, shape.getDims().data(), sizeof(u32) * shape.dimCnt(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(mds + shape.dimCnt(), shape.getStrides().data(), sizeof(u32) * shape.dimCnt(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemset(mdst, 0, sizeof(f64) * shape.tensorSize() / shape.getDim(d)), "cuda_memset");

    if (d == shape.dimCnt() - 1) {
        u32 gridy_dim = (shape.tensorSize() / shape.getDim(shape.dimCnt() - 1) + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        u32 gridx_dim = (shape.getDim(shape.dimCnt() - 1) + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        cuda_kernel::sum<<<dim3(gridx_dim, gridy_dim), dim3(sqrt_tcnt_per_block(), sqrt_tcnt_per_block())>>>(msrc, mdst, mds, mds + shape.dimCnt(), shape.dimCnt());
        CHECK_CUDA_CALL(hipPeekAtLastError(), "sum1");
    } else {
        u32 gridy_dim = (shape.getStrides()[d] + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        u32 gridz_dim = (shape.getDim(d) + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        u32 gridx_dim = shape.tensorSize() / shape.getStrides()[d] / shape.getDim(d);
        cuda_kernel::sum<<<dim3(gridx_dim, gridy_dim, gridz_dim), dim3(1, sqrt_tcnt_per_block(), sqrt_tcnt_per_block())>>>(msrc, mdst, mds, mds + shape.dimCnt(), shape.dimCnt(), d);
        CHECK_CUDA_CALL(hipPeekAtLastError(), "sum2");
    }
    CHECK_CUDA_CALL(hipMemcpy(dst, mdst, sizeof(f64) * shape.tensorSize() / shape.getDim(d), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

void expand(const f64* src, f64* dst, const base::Shape&, u32 d, u32 expd) {

}

void transpose(f64* data, const base::Shape& shape, u32 d1, u32 d2) {
    if (d1 > d2) {std::swap(d1, d2);}

    auto transpose_shape = shape.transpose(d1, d2);

    f64 *msrc, *mdst;
    u32 *mds;
    CHECK_MALLOC(Mem::malloc((void**)&msrc, sizeof(f64) * shape.tensorSize()));
    utils::Defer free_msrc([&msrc]() {Mem::free(msrc);});
    CHECK_MALLOC(Mem::malloc((void**)&mdst, sizeof(f64) * transpose_shape.tensorSize()));
    utils::Defer free_mdst([&mdst]() {Mem::free(mdst);});
    CHECK_MALLOC(Mem::malloc((void**)&mds, sizeof(u32) * shape.dimCnt() * 3));
    utils::Defer free_mds([&mds]() {Mem::free(mds);});

    CHECK_CUDA_CALL(hipMemcpy(msrc, data, sizeof(f64) * shape.tensorSize(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(mds, shape.getDims().data(), sizeof(u32) * shape.dimCnt(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(mds + shape.dimCnt(), shape.getStrides().data(), sizeof(u32) * shape.dimCnt(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(mds + shape.dimCnt() * 2, transpose_shape.getStrides().data(), sizeof(u32) * shape.dimCnt(), hipMemcpyHostToDevice), "cuda_memcpy_h2d");

    if (d2 == shape.dimCnt() - 1) {
        u32 gridy_dim = (shape.tensorSize() / shape.getStrides()[d1] + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        u32 gridx_dim = (shape.getStrides()[d1] + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        cuda_kernel::transpose<<<dim3(gridx_dim, gridy_dim), dim3(sqrt_tcnt_per_block(), sqrt_tcnt_per_block())>>>(msrc, mdst, mds, mds + shape.dimCnt(), mds + shape.dimCnt() * 2, shape.dimCnt(), d1);
        CHECK_CUDA_CALL(hipPeekAtLastError(), "tranpose1");
    } else {
        u32 grid_dim = (shape.tensorSize() + sqrt_tcnt_per_block() - 1) / sqrt_tcnt_per_block();
        cuda_kernel::transpose<<<dim3(grid_dim), dim3(sqrt_tcnt_per_block())>>>(msrc, mdst, mds, mds + shape.dimCnt(), mds + shape.dimCnt() * 2, shape.dimCnt(), d1, d2);
        CHECK_CUDA_CALL(hipPeekAtLastError(), "tranpose2");
    }

    CHECK_CUDA_CALL(hipMemcpy(data, mdst, sizeof(f64) * shape.tensorSize(), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

void mmul(u32 m, u32 n, u32 k, const f64* data1, const f64* data2, f64* dst, u32 size) {
    hipblasHandle_t handle; 
    CHECK_CUBLAS_CALL(hipblasCreate(&handle), "create"); 
    utils::Defer destroy_handle([&handle]() {hipblasDestroy(handle);});

    f64 *cm, **icm, **hicm;
    CHECK_MALLOC(Mem::malloc((void**)&cm, (m * k + k * n + m * n) * size * sizeof(f64)));
    utils::Defer destroy_cm([&cm]() {Mem::free(cm);});
    CHECK_MALLOC(Mem::malloc((void**)&icm, 3 * size * sizeof(f64*)));
    utils::Defer destroy_icm([&icm]() {Mem::free(icm);});
    hicm = (f64**)malloc(3 * size * sizeof(f64*));
    utils::Defer destroy_hicm([&hicm]() {free(hicm);});

    CHECK_CUDA_CALL(hipMemcpy(cm, data1, sizeof(f64) * m * k * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    CHECK_CUDA_CALL(hipMemcpy(cm + m * k * size, data2, sizeof(f64) * k * n * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    // dst不需要拷贝

    for (u32 i = 0; i < size; i++) {
        hicm[i] = cm + i * m * k;
        hicm[i + size] = cm + size * m * k + i * k * n;
        hicm[i + 2 * size] = cm + size * m * k + size * k * n + i * m * n;
    }
    CHECK_CUDA_CALL(hipMemcpy(icm, hicm, 3 * size * sizeof(f64*), hipMemcpyHostToDevice), "cuda_memcpy_h2d");

    const f64 alpha = 1.0, beta = 0.0; 
    CHECK_CUBLAS_CALL(hipblasDgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, icm + size, n, icm, k, &beta, icm + 2 * size, n, size), "gemm_batched");

    CHECK_CUDA_CALL(hipMemcpy(dst, cm + m * k * size + k * n * size, sizeof(f64) * m * n * size, hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
}

bool inv(u32 m, f64* data, u32 size) {
    hipblasHandle_t handle; 
    CHECK_CUBLAS_CALL(hipblasCreate(&handle), "create"); 
    utils::Defer destroy_handle([&handle]() {hipblasDestroy(handle);});

    f64 *cm, **icm, **hicm;
    CHECK_MALLOC(Mem::malloc((void**)&cm, 2 * m * m * size * sizeof(f64)));
    utils::Defer destroy_cm([&cm]() {Mem::free(cm);});
    CHECK_MALLOC(Mem::malloc((void**)&icm, 2 * size * sizeof(f64*)));
    utils::Defer destroy_icm([&icm]() {Mem::free(icm);});
    hicm = (f64**)malloc(2 * size * sizeof(f64*));
    utils::Defer destroy_hicm([&hicm]() {free(hicm);});

    CHECK_CUDA_CALL(hipMemcpy(cm, data, sizeof(f64) * m * m * size, hipMemcpyHostToDevice), "cuda_memcpy_h2d");
    for (u32 i = 0; i < 2 * size; i++) {
        hicm[i] = cm + i * m * m;
    }
    CHECK_CUDA_CALL(hipMemcpy(icm, hicm, 2 * size * sizeof(f64*), hipMemcpyHostToDevice), "cuda_memcpy_h2d");

    int *pivot_arr, *info_arr, *hinfo_arr;
    CHECK_MALLOC(Mem::malloc((void**)&pivot_arr, m * size * sizeof(int)));
    utils::Defer destroy_pivot_arr([&pivot_arr]() {Mem::free(pivot_arr);});
    CHECK_MALLOC(Mem::malloc((void**)&info_arr, size * sizeof(int)));
    utils::Defer destroy_info_arr([&info_arr]() {Mem::free(info_arr);});
    hinfo_arr = (int*)malloc(size * sizeof(int));
    utils::Defer destroy_hinfo_arr([&hinfo_arr]() {free(hinfo_arr);});

    CHECK_CUBLAS_CALL(hipblasDgetrfBatched(handle, m, icm, m, pivot_arr, info_arr, size), "getrf_batched");

    auto check_info = [&hinfo_arr, &info_arr, &size](const std::string& fn) -> bool {
        CHECK_CUDA_CALL(hipMemcpy(hinfo_arr, info_arr, size * sizeof(int), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
        for (u32 i = 0; i < size; i++) {
            if (hinfo_arr[i] != 0) {
                LOG(ERROR) << __FUNCTION__ << " check info err: " << fn << " index: " << i << " code: " << hinfo_arr[i];
                return false;
            }
        }
        return true;
    };
    if (!check_info("getrf_batched")) {
        return false;
    }

    CHECK_CUBLAS_CALL(hipblasDgetriBatched(handle, m, icm, m, pivot_arr, icm + size, m, info_arr, size), "dgetri_batched");
    if (!check_info("getri_batched")) {
        return false;
    }

    CHECK_CUDA_CALL(hipMemcpy(data, cm + m * m * size, m * m * size * sizeof(f64), hipMemcpyDeviceToHost), "cuda_memcpy_d2h");
    return true;
}




}